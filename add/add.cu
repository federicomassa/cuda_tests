
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  std::cout << __LINE__ << std::endl;
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

    std::cout << __LINE__ << std::endl;

    if (!x || !y) {
      std::cout << "WTF" << std::endl;
      exit(1);
    }
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    std::cout << "HEY" << std::endl;
    std::cout << i << std::endl;
    std::cout << N << std::endl;
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

    std::cout << __LINE__ << std::endl;
  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

    std::cout << __LINE__ << std::endl;
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

    std::cout << __LINE__ << std::endl;
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

    std::cout << __LINE__ << std::endl;
  // Free memory
  hipFree(x);
  hipFree(y);

    std::cout << __LINE__ << std::endl;
  
  return 0;
}
  
