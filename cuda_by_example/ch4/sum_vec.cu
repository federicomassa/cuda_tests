#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <iostream>
#include <chrono>
#include <sstream>

long N = 20000000;
int THREADS = 256;
int BLOCKS = std::ceil(double(N)/double(THREADS));

void add(int* a, int* b, int* c) {
	int tid = 0;
	
	while (tid < N) {
		c[tid] =  a[tid] + b[tid];
		tid++;
	}
}

__global__
void add_gpu(int* a, int* b, int* c, long* N) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < *N) { 
		c[tid] =  a[tid] + b[tid];
		tid += blockDim.x*gridDim.x;
	}
}

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::milliseconds ms;
typedef std::chrono::duration<float> fsec;

void cpu_test() {
	int* a = new int[N];
	int* b = new int[N];
	int* c = new int[N];
	
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}
	
	printf("Starting CPU benchmark...\n");
	
	auto t0 = Time::now();
	add(a,b,c);
	auto t1 = Time::now();
	
	
    fsec fs = t1 - t0;
	std::cout << "CPU took: " << fs.count() << " s\n";	

	delete a, b, c;
}

void gpu_test() {
	printf("Starting GPU benchmark...\n");
	
	long* dev_N;
	hipMalloc((void**)&dev_N, sizeof(long));
	hipMemcpy(dev_N, &N, sizeof(long), hipMemcpyHostToDevice);
	
	int* a = new int[N];
	int* b = new int[N];
	int* c = new int[N];
	
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}
	
	int* dev_a, *dev_b, *dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice));
	
	// Add and transfer result to CPU
	auto t0 = Time::now();

	add_gpu<<<THREADS,THREADS>>>(dev_a, dev_b, dev_c, dev_N);
	
	auto t1 = Time::now();
	
	HANDLE_ERROR(hipMemcpy(a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(b, dev_b, N*sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));
	auto t2 = Time::now();
	
	fsec fs = t1 - t0;
	fsec fs2 = t2 - t1;

	printf("GPU took %f s (%f to sum + %f to retrieve data from device)", (fs + fs2).count(), fs.count(), fs2.count());
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_N);
	
	delete a,b,c;
}

int main(int argc, char** argv) {
	std::cout << "----------- Summing vectors of size " << N << "-----------" << std::endl;

	cpu_test();
	
	printf("##################################\n\n");
		
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		
		printf("Starting GPU benchmark on device %d with name: %s\n", i, prop.name);
		hipSetDevice(i);
		
		gpu_test();
		printf("\n###############################\n");
	}
	
	return 0;
}
