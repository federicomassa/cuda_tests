#include "hip/hip_runtime.h"
#include "../common/book.h"
#include <iostream>
#include <chrono>
#include <sstream>

const long N = 10000000;
const int THREADS = 512;
const int BLOCKS = std::ceil(double(N)/double(THREADS));
//const int BLOCKS = 512;

void add(int* a, int* b, int* c) {
	int tid = 0;
	
	while (tid < N) {
	  c[tid] =  a[tid] + b[tid];
	  tid++;
	}
}

__global__
void add_gpu(int* a, int* b, int* c) {
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	while (tid < N) {
	  c[tid] =  a[tid] + b[tid];
	  tid += blockDim.x*gridDim.x;
	}
}

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::milliseconds ms;
typedef std::chrono::duration<float> fsec;

void cpu_test() {
	int* a = new int[N];
	int* b = new int[N];
	int* c = new int[N];
	
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i+2;
	}
	
	printf("Starting CPU benchmark...\n");
	
	auto t0 = Time::now();
	add(a,b,c);
	auto t1 = Time::now();
	
	
	fsec fs = t1 - t0;
	std::cout << "CPU took: " << fs.count() << " s\n";
	std::cout << "Test: central index is: " << c[N/2] << std::endl;

	delete a, b, c;
}

void gpu_test() {
	printf("Starting GPU benchmark...\n");
	
	int* a = new int[N];
	int* b = new int[N];
	int* c = new int[N];
	
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i+2;
	}
	
	int* dev_a, *dev_b, *dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N*sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));
	
	// Add and transfer result to CPU
	auto t0 = Time::now();

	add_gpu<<<BLOCKS,THREADS>>>(dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	auto t1 = Time::now();
	
	HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));
	auto t2 = Time::now();
	
	fsec fs = t1 - t0;
	fsec fs2 = t2 - t1;

	printf("GPU took %f s (%f to sum + %f to retrieve data from device)\n", (fs + fs2).count(), fs.count(), fs2.count());
	std::cout << "Test: central index is: " << c[N/2] << std::endl;
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	delete a,b,c;
}

int main(int argc, char** argv) {
	std::cout << "----------- Summing vectors of size " << N << "-----------" << std::endl;

	cpu_test();
	
	printf("##################################\n\n");
		
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	
	for (int i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		
		printf("Starting GPU benchmark on device %d with name: %s\n", i, prop.name);
		hipSetDevice(i);
		
		gpu_test();
		printf("\n###############################\n");
	}
	
	return 0;
}
